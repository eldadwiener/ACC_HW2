#include "hip/hip_runtime.h"
/* compile with: nvcc -O3 -maxrregcount=32 hw2.cu -o hw2 */

#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <time.h>
#include <assert.h>
#include <string.h>

///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
#define IMG_DIMENSION 32
#define NREQUESTS 10000
#define DONEJOB -1
typedef unsigned char uchar;

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

#define SQR(a) ((a) * (a))

void process_image(uchar *img_in, uchar *img_out) {
    int histogram[256] = { 0 };
    for (int i = 0; i < SQR(IMG_DIMENSION); i++) {
        histogram[img_in[i]]++;
    }

    int cdf[256] = { 0 };
    int hist_sum = 0;
    for (int i = 0; i < 256; i++) {
        hist_sum += histogram[i];
        cdf[i] = hist_sum;
    }

    int cdf_min = 0;
    for (int i = 0; i < 256; i++) {
        if (cdf[i] != 0) {
            cdf_min = cdf[i];
            break;
        }
    }

    uchar map[256] = { 0 };
    for (int i = 0; i < 256; i++) {
        int map_value = (float)(cdf[i] - cdf_min) / (SQR(IMG_DIMENSION) - cdf_min) * 255;
        map[i] = (uchar)map_value;
    }

    for (int i = 0; i < SQR(IMG_DIMENSION); i++) {
        img_out[i] = map[img_in[i]];
    }
}

double static inline get_time_msec(void) {
    struct timespec t;
    int res = clock_gettime(CLOCK_MONOTONIC, &t);
    if (res) {
        perror("clock_gettime failed");
        exit(1);
    }
    return t.tv_sec * 1e+3 + t.tv_nsec * 1e-6;
}

/* we'll use these to rate limit the request load */
struct rate_limit_t {
    double last_checked;
    double lambda;
    unsigned seed;
};


void rate_limit_init(struct rate_limit_t *rate_limit, double lambda, int seed) {
    rate_limit->lambda = lambda;
    rate_limit->seed = (seed == -1) ? 0 : seed;
    rate_limit->last_checked = 0;
}

int rate_limit_can_send(struct rate_limit_t *rate_limit) {
    if (rate_limit->lambda == 0) return 1;
    double now = get_time_msec() * 1e-3;
    double dt = now - rate_limit->last_checked;
    double p = dt * rate_limit->lambda;
    rate_limit->last_checked = now;
    if (p > 1) p = 1;
    double r = (double)rand_r(&rate_limit->seed) / RAND_MAX;
    return (p > r);
}

double distance_sqr_between_image_arrays(uchar *img_arr1, uchar *img_arr2) {
    double distance_sqr = 0;
    for (int i = 0; i < NREQUESTS * SQR(IMG_DIMENSION); i++) {
        distance_sqr += SQR(img_arr1[i] - img_arr2[i]);
    }
    return distance_sqr;
}

/* we won't load actual files. just fill the images with random bytes */
void load_images(uchar *images) {
    srand(0);
    for (int i = 0; i < NREQUESTS * SQR(IMG_DIMENSION); i++) {
        images[i] = rand() % 256;
    }
}

__device__ int arr_min(int arr[], int arr_size) {
    // we assume arr_size threads call this function for arr[]
    __shared__ int SharedMin;
    int tid = threadIdx.x;
    for(int stride = 0; stride < arr_size; stride += blockDim.x)
    {
        if( (tid + stride < arr_size) && 
            (arr[tid + stride] > 0) && 
            ((tid + stride == 0) || (arr[tid + stride - 1] == 0))) // cdf is a rising function, so only the first non zero will have zero before it.
        {
            SharedMin = arr[tid + stride];
        }
        __syncthreads();
    }
    return SharedMin;
}

__device__ int arr_min_ref(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int rhs, lhs;

    for (int stride = 1; stride < arr_size; stride *= 2) {
        if (tid >= stride && tid < arr_size) {
            rhs = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride && tid < arr_size) {
            lhs = arr[tid];
            if (rhs != 0) {
                if (lhs == 0)
                    arr[tid] = rhs;
                else
                    arr[tid] = min(arr[tid], rhs);
            }
        }
        __syncthreads();
    }

    int ret = arr[arr_size - 1];
    return ret;
}

__device__ void prefix_sum(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int increment;

    for (int stride = 1; stride < min(blockDim.x, arr_size); stride *= 2) {
        if (tid >= stride && tid < arr_size) {
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride && tid < arr_size) {
            arr[tid] += increment;
        }
        __syncthreads();
    }
}

__global__ void gpu_process_image(uchar *in, uchar *out) {
    __shared__ int histogram[256];
    __shared__ int hist_min[256];

    int tid = threadIdx.x;

    if (tid < 256) {
        histogram[tid] = 0;
    }
    __syncthreads();

    for (int i = tid; i < SQR(IMG_DIMENSION); i += blockDim.x)
        atomicAdd(&histogram[in[i]], 1);

    __syncthreads();

    prefix_sum(histogram, 256);

    if (tid < 256) {
        hist_min[tid] = histogram[tid];
    }
    __syncthreads();

    int cdf_min = arr_min(hist_min, 256);

    __shared__ uchar map[256];
    if (tid < 256) {
        int map_value = (float)(histogram[tid] - cdf_min) / (SQR(IMG_DIMENSION) - cdf_min) * 255;
        map[tid] = (uchar)map_value;
    }

    __syncthreads();

    for (int i = tid; i < SQR(IMG_DIMENSION); i += blockDim.x) {
        out[i] = map[in[i]];
    }
    return;
}

void print_usage_and_die(char *progname) {
    printf("usage:\n");
    printf("%s streams <load (requests/sec)>\n", progname);
    printf("OR\n");
    printf("%s queue <#threads> <load (requests/sec)>\n", progname);
    exit(1);
}

int _min(int a, int b) {
    return (a<b) ? a : b;
}

unsigned int getTBlocksAmnt(int threadsPerBlock, int shmemPerBlock) {
    struct hipDeviceProp_t props;
    CUDA_CHECK( hipGetDeviceProperties(&props, 0) );
    int  ThreadsPerSM = min(props.maxThreadsPerMultiProcessor, props.regsPerMultiprocessor/32);
    int  SMCount = props.multiProcessorCount;
    size_t  shmemPerSM = props.sharedMemPerMultiprocessor;
    return SMCount * min( ThreadsPerSM/threadsPerBlock, (unsigned int)shmemPerSM/shmemPerBlock);
}

#define QSIZE 10
typedef struct QmetaData {
    int tail;
    int head;
    int size;
    bool done;
} QmetaData;

typedef struct jobS* pJobS;
typedef struct jobS {
    uchar job[SQR(IMG_DIMENSION)];
    int jobId;
} jobS;

//TODO maybe remove?
typedef struct singleQ {
    jobS jobs[QSIZE];
    int head;
    int tail;
} Q;

typedef struct pcQ {
    QmetaData* meta;
    pJobS queue;
    int* usedCells;
}pcQ;

typedef struct tbMem {
    uchar memIn[SQR(IMG_DIMENSION)];
    uchar memOut[SQR(IMG_DIMENSION)];
} tbMem;

__host__ void setQ(pcQ& queue, void* allocated, unsigned int Qsize) {
    queue.meta = (QmetaData*)allocated;
    queue.meta->size = Qsize;
    queue.meta->head = 0;
    queue.meta->tail = 0;
    queue.meta->done = false;
    queue.queue = (pJobS)(queue.meta + 1);
    queue.usedCells = (int*)( queue.queue + queue.meta->size );
    for (int i = 0; i < queue.meta->size; ++i ){
        queue.usedCells[i] = 0;
    }
} 

__device__ void setQ(pcQ& queue, void* allocated) {
    queue.meta = (QmetaData*)allocated;
    queue.queue = (pJobS)(queue.meta + 1);
    queue.usedCells = (int*)( queue.queue + queue.meta->size );
}


__global__ void gpu_process_image_pc(void* in,void* out, tbMem* tb_mem) {
    __shared__ int histogram[256];
    __shared__ int hist_min[256];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    // parse given pointers into useful structs
    pcQ inQ, outQ;
    setQ(inQ, in);
    setQ(outQ, out);
    tbMem currJob = tb_mem[bid];
    uchar * jobQptr;
    int* jobUsedCellPtr;
    unsigned int currJobId;
    int QbuffNum = inQ.meta->size;
    while (true){ 
        // if Q is empty and cpu is done, exit the kernel
        //try to catch a buffer
        __shared__ bool stop;// TODO add to threadblock amnt calculation
        stop = false;
        while (tid == 0){
            __threadfence();
            while(atomicCAS(&(inQ.usedCells[inQ.meta->tail % QbuffNum]), 1 ,2) != 1)
            {
                if(inQ.meta->tail == inQ.meta->head && inQ.meta->done) {
                    stop = true;
                    break;
                }
            }
            __threadfence();
            if(stop) break;
            int idx = inQ.meta->tail % QbuffNum;
            //save the job ptr and the job id
            jobQptr = inQ.queue[inQ.meta->tail % QbuffNum].job;
            currJobId = inQ.queue[inQ.meta->tail % QbuffNum].jobId;
            jobUsedCellPtr = inQ.usedCells + (inQ.meta->tail % QbuffNum);
            //move the tail forward to allow athoer T.Bs to work
            atomicAdd(&inQ.meta->tail,1);
            /* ---------TODO: do this copy mor efficient---------------------------------*/
            memcpy(currJob.memIn, jobQptr, SQR(IMG_DIMENSION));
            __threadfence();
            //*jobUsedCellPtr = 0; //the cell is empty now
            inQ.usedCells[idx] = 0;
            /*----------------------------------------------------------------------------*/
            break;
        }
        __syncthreads(); //wait for thread 0 to catch a job
        if (stop)
            return;
        /*do here the copy*/
        //
        //
        //do the calcs
        if (tid < 256) {
            histogram[tid] = 0;
        }
        __syncthreads();
    
        for (int i = tid; i < SQR(IMG_DIMENSION); i += blockDim.x)
            atomicAdd(&histogram[currJob.memIn[i]], 1);
    
        __syncthreads();
    
        prefix_sum(histogram, 256);
    
        if (tid < 256) {
            hist_min[tid] = histogram[tid];
        }
        __syncthreads();
    
        int cdf_min = arr_min(hist_min, 256);
    
        __shared__ uchar map[256];
        if (tid < 256) {
            int map_value = (float)(histogram[tid] - cdf_min) / (SQR(IMG_DIMENSION) - cdf_min) * 255;
            map[tid] = (uchar)map_value;
        }
    
        __syncthreads();
    
        for (int i = tid; i < SQR(IMG_DIMENSION); i += blockDim.x) {
            currJob.memOut[i] = map[currJob.memIn[i]];
        }
        // try to catch free cell in Qout and copy the result
        if (tid == 0){
            while(atomicCAS(&(outQ.usedCells[outQ.meta->head % QbuffNum]), 0 ,2) != 0);
            //save the job-out ptr and insert the job id
            jobQptr = outQ.queue[outQ.meta->head % QbuffNum].job;
            printf("GPU sending job #%d\n",outQ.queue[outQ.meta->head % QbuffNum].jobId); // TODO REMOVE
            outQ.queue[outQ.meta->head % QbuffNum].jobId = currJobId;
            jobUsedCellPtr = outQ.usedCells + (outQ.meta->head % QbuffNum);
            outQ.meta->head ++;
            /* ---------TODO: do this copy mor efficient---------------------------------*/
            memcpy(jobQptr, currJob.memOut, SQR(IMG_DIMENSION));
            *jobUsedCellPtr = 1; //the cell is ready for read now
            /*----------------------------------------------------------------------------*/
        }
    }
}


__global__ void gpu_process_image_pc2(volatile void* in,volatile void* out) {
    __shared__ int histogram[256];
    __shared__ int hist_min[256];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    uchar * jobQptr;
    int currJobId;
    Q *inQ = (Q*)in, *outQ = (Q*)out;
    inQ += bid;
    outQ += bid;
    while (true) {
        __threadfence();
        while(inQ->tail >= inQ->head) __threadfence_system(); // wait for queue to contain a job
        //save the job ptr and the job id
        jobQptr = inQ->jobs[inQ->tail % QSIZE].job;
        currJobId = inQ->jobs[inQ->tail % QSIZE].jobId;
        if(currJobId == DONEJOB) {
            __threadfence();
            return;
        }
        /*----------------------------------------------------------------------------*/
        /*do here the copy*/
        //
        //
        //do the calcs

        __threadfence();

        if (tid < 256) {
            histogram[tid] = 0;
        }
        __syncthreads();
    
        for (int i = tid; i < SQR(IMG_DIMENSION); i += blockDim.x)
        {
            __threadfence_system(); // needed to get currect image data
            atomicAdd(&histogram[jobQptr[i]], 1);
        }
    
        __syncthreads();
    
        prefix_sum(histogram, 256);
    
        if (tid < 256) {
            hist_min[tid] = histogram[tid];
        }
        __syncthreads();
    
        int cdf_min = arr_min(hist_min, 256);
    
        __shared__ uchar map[256];
        if (tid < 256) {
            int map_value = (float)(histogram[tid] - cdf_min) / (SQR(IMG_DIMENSION) - cdf_min) * 255;
            map[tid] = (uchar)map_value;
        }
    
        __syncthreads();
        while(!(outQ->head - outQ->tail < QSIZE)) __threadfence_system();
        for (int i = tid; i < SQR(IMG_DIMENSION); i += blockDim.x) {
            outQ->jobs[outQ->head % QSIZE].job[i] = map[jobQptr[i]];
        }
        outQ->jobs[outQ->head % QSIZE].jobId = currJobId; 
        __threadfence_system();
        // try to catch free cell in Qout and copy the result
        if (tid == 0){
            //save the job-out ptr and insert the job id
            outQ->head ++;
            inQ->tail ++;
            // printf("GPU: sent job #%d\n",currJobId);
        }
        __threadfence_system();
    }
}

enum {PROGRAM_MODE_STREAMS = 0, PROGRAM_MODE_QUEUE};
int main(int argc, char *argv[]) {

    int mode = -1;
    int threads_queue_mode = -1; /* valid only when mode = queue */
    double load = 0;
    if (argc < 3) print_usage_and_die(argv[0]);

    if (!strcmp(argv[1], "streams")) {
        if (argc != 3) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_STREAMS;
        load = atof(argv[2]);
    } else if (!strcmp(argv[1], "queue")) {
        if (argc != 4) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_QUEUE;
        threads_queue_mode = atoi(argv[2]);
        load = atof(argv[3]);
    } else {
        print_usage_and_die(argv[0]);
    }

    uchar *images_in; /* we concatenate all images in one huge array */
    uchar *images_out;
    CUDA_CHECK( hipHostAlloc(&images_in, NREQUESTS * SQR(IMG_DIMENSION), 0) );
    CUDA_CHECK( hipHostAlloc(&images_out, NREQUESTS * SQR(IMG_DIMENSION), 0) );

    load_images(images_in);
    double t_start, t_finish;

    /* using CPU */
    printf("\n=== CPU ===\n");
    t_start  = get_time_msec();
    for (int img_idx = 0; img_idx < NREQUESTS; ++img_idx)
        process_image(&images_in[img_idx * SQR(IMG_DIMENSION)], &images_out[img_idx * SQR(IMG_DIMENSION)]);
    t_finish = get_time_msec();
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);

    double total_distance = 0;

    /* using GPU task-serial.. just to verify the GPU code makes sense */
    printf("\n=== GPU Task Serial ===\n");

    uchar *images_out_from_gpu;
    CUDA_CHECK( hipHostAlloc(&images_out_from_gpu, NREQUESTS * SQR(IMG_DIMENSION), 0) );

    do {
        uchar *gpu_image_in, *gpu_image_out;
        CUDA_CHECK(hipMalloc(&gpu_image_in, SQR(IMG_DIMENSION)));
        CUDA_CHECK(hipMalloc(&gpu_image_out, SQR(IMG_DIMENSION)));

        t_start = get_time_msec();
        for (int img_idx = 0; img_idx < NREQUESTS; ++img_idx) {
            CUDA_CHECK(hipMemcpy(gpu_image_in, &images_in[img_idx * SQR(IMG_DIMENSION)], SQR(IMG_DIMENSION), hipMemcpyHostToDevice));
            gpu_process_image<<<1, 1024>>>(gpu_image_in, gpu_image_out);
            CUDA_CHECK(hipMemcpy(&images_out_from_gpu[img_idx * SQR(IMG_DIMENSION)], gpu_image_out, SQR(IMG_DIMENSION), hipMemcpyDeviceToHost));
        }
        total_distance += distance_sqr_between_image_arrays(images_out, images_out_from_gpu);
        CUDA_CHECK(hipDeviceSynchronize());
        t_finish = get_time_msec();
        printf("distance from baseline %lf (should be zero)\n", total_distance);
        printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);

        CUDA_CHECK(hipFree(gpu_image_in));
        CUDA_CHECK(hipFree(gpu_image_out));
    } while (0);

    /* now for the client-server part */
    printf("\n=== Client-Server ===\n");
    double *req_t_start = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_start, 0, NREQUESTS * sizeof(double));

    double *req_t_end = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_end, 0, NREQUESTS * sizeof(double));

    struct rate_limit_t rate_limit;
    rate_limit_init(&rate_limit, load, 0);

    /* TODO allocate / initialize memory, streams, etc... */
    hipStream_t streams[64];
    int imgInStream[64];
    uchar *gpu_image_in[64], *gpu_image_out[64];
    for(int i = 0; i < 64; i++) {
        hipStreamCreate(&streams[i]);
        imgInStream[i] = -1;
        CUDA_CHECK(hipMalloc(&gpu_image_in[i], SQR(IMG_DIMENSION)));
        CUDA_CHECK(hipMalloc(&gpu_image_out[i], SQR(IMG_DIMENSION)));
    }
    CUDA_CHECK(hipMemset(images_out_from_gpu, 0, NREQUESTS * SQR(IMG_DIMENSION)));

    double ti = get_time_msec();
    if (mode == PROGRAM_MODE_STREAMS) {
        for (int img_idx = 0; img_idx < NREQUESTS; ++img_idx) {

            /* TODO query (don't block) streams for any completed requests.
             * update req_t_end of completed requests
             */
            int availStream = -1;
            for(int i=0; i < 64; ++i)
            {
                if(hipStreamQuery(streams[i]) == hipSuccess)
                {
                    if(availStream == -1)
                        availStream = i;
                    if(imgInStream[i] > -1)
                    {    
                        req_t_end[imgInStream[i]] = get_time_msec();
                        //printf("Img num: %d, start time: %f, end time: %f\n",imgInStream[i],req_t_start[imgInStream[i]],req_t_end[imgInStream[i]]); // REMOVE DEBUG
                        imgInStream[i] = -1;
                    }
                }
            }
            if (availStream == -1 || !rate_limit_can_send(&rate_limit)) {
                --img_idx;
                continue;
            }
            //printf("Sending img id: %d to stream id: %d\n",img_idx, availStream); // REMOVE DEBUG
            imgInStream[availStream] = img_idx;
            req_t_start[img_idx] = get_time_msec();

            /* TODO place memcpy's and kernels in a stream */
            hipMemcpyAsync(gpu_image_in[availStream], &images_in[img_idx * SQR(IMG_DIMENSION)], SQR(IMG_DIMENSION),hipMemcpyHostToDevice,streams[availStream]);
            gpu_process_image<<<1, SQR(IMG_DIMENSION) ,0, streams[availStream]>>>(gpu_image_in[availStream], gpu_image_out[availStream]);
            hipMemcpyAsync(&images_out_from_gpu[img_idx * SQR(IMG_DIMENSION)], gpu_image_out[availStream], SQR(IMG_DIMENSION),hipMemcpyDeviceToHost,streams[availStream]);
        }
        /* TODO now make sure to wait for all streams to finish */
        hipDeviceSynchronize();
        // Mark all remaining images end time to now
        double endTime = get_time_msec();
        for(int i = 0; i < 64 ; ++i)
        {
            if(imgInStream[i] > -1)
            {
                req_t_end[imgInStream[i]] = endTime;
            }
        }
        //TODO, maybe need to move mem free further down
        for(int i = 0; i < 64; ++i)
        {
        CUDA_CHECK(hipStreamDestroy(streams[i]));    
        CUDA_CHECK(hipFree(gpu_image_in[i]));
        CUDA_CHECK(hipFree(gpu_image_out[i]));
        }

    } else if (mode == PROGRAM_MODE_QUEUE) {
        // TODO launch GPU consumer-producer kernel
        unsigned int tblocks = getTBlocksAmnt(threads_queue_mode, 2*4*256+256+4);
        unsigned int amntRecv = 0;
        unsigned int nextIns = 0; // first candidate block for next img insert, to implement RR
        Q *QinHost, *QinDev;
        Q *QoutHost, *QoutDev;
        CUDA_CHECK( hipHostAlloc(&QinHost, sizeof(Q)*tblocks , 0) );
        CUDA_CHECK( hipHostAlloc(&QoutHost, sizeof(Q)*tblocks , 0) );
        memset(QinHost, 0, sizeof(Q)*tblocks);
        memset(QoutHost, 0, sizeof(Q)*tblocks);
        CUDA_CHECK( hipHostGetDevicePointer(&QinDev, QinHost, 0) );
        CUDA_CHECK( hipHostGetDevicePointer(&QoutDev, QoutHost, 0) );
        gpu_process_image_pc2<<<tblocks, threads_queue_mode>>>(QinDev, QoutDev);
        __sync_synchronize();
        for (int img_idx = 0; img_idx < NREQUESTS; ++img_idx) {
            __sync_synchronize();
            // check each tblock's out queue
            for (int block = 0; block < tblocks; ++block)
            {
                __sync_synchronize();
                while ( QoutHost[block].tail < QoutHost[block].head )
                {
                    __sync_synchronize();
                    jobS& job = QoutHost[block].jobs[QoutHost[block].tail % QSIZE ];
                    // __sync_synchronize();
                    memcpy(images_out_from_gpu + (job.jobId * SQR(IMG_DIMENSION)), job.job, SQR(IMG_DIMENSION) );
                    // __sync_synchronize();
                    QoutHost[block].tail++;
                    __sync_synchronize();
                    req_t_end[job.jobId] = get_time_msec();
                    ++amntRecv;
                    // printf("received completed job #%d\n",job.jobId);
                }
            }
            // Check send rate limit
            if (!rate_limit_can_send(&rate_limit))
            {
                --img_idx;
                continue;
            }
            //printf("checking if theres room for img: %d\n",img_idx);
            int blockToUse;
            bool failed = true;
            for (int i = 0; i < tblocks; ++i)
            {
                blockToUse = (i + nextIns) % tblocks;
                //__sync_synchronize();
                if( QinHost[blockToUse].tail + QSIZE <= QinHost[blockToUse].head ) //the next cell isn't empty
                {
                    continue;
                } 
                //__sync_synchronize();
                failed = false;
                break;
            }
            if ( failed )
            {
                img_idx--;
                continue;
            }
            __sync_synchronize();
            req_t_start[img_idx] = get_time_msec();
            // TODO push task to queue 
            // printf("pushing img #%d, to threadblock #%d\n",img_idx, blockToUse);
            __sync_synchronize();
            jobS &inJob = QinHost[blockToUse].jobs[QinHost[blockToUse].head % QSIZE];
            __sync_synchronize();
            inJob.jobId = img_idx;
            __sync_synchronize();
            memcpy(inJob.job, images_in + (img_idx * SQR(IMG_DIMENSION)), SQR(IMG_DIMENSION));
            __sync_synchronize();
            QinHost[blockToUse].head ++;
            __sync_synchronize();

            ++nextIns;
        }
        // done, signal all threads to finish and stop 
        for (int block = 0; block < tblocks; ++block)
        {
            while(QinHost[block].tail + QSIZE <= QinHost[block].head) __sync_synchronize();
            QinHost[block].jobs[QinHost[block].head % QSIZE].jobId = DONEJOB;
            __sync_synchronize();
            ++QinHost[block].head;
            __sync_synchronize();
        }
        // printf("syncing\n");
        __sync_synchronize();
        // hipDeviceSynchronize(); // TODO DO WE NEED THIS
        // printf("done syncing\n");
        // get the rest of the images to cpu
        // printf("So far received: %d jobs\n",amntRecv);
        while(amntRecv < NREQUESTS) {
            hipDeviceSynchronize();
            __sync_synchronize();
            for (int block = 0; block < tblocks; ++block)
            {
                __sync_synchronize();
                while ( QoutHost[block].tail < QoutHost[block].head )
                {
                    __sync_synchronize();
                    jobS& job = QoutHost[block].jobs[QoutHost[block].tail % QSIZE ];
                    __sync_synchronize();
                    memcpy(images_out_from_gpu + (job.jobId * SQR(IMG_DIMENSION)), job.job, SQR(IMG_DIMENSION) );
                    __sync_synchronize();
                    QoutHost[block].tail++;
                    __sync_synchronize();
                    req_t_end[job.jobId] = get_time_msec();
                    ++amntRecv;
                    // printf("received completed job #%d\n",job.jobId);
                }
            }
        }
        __sync_synchronize();
        printf("In total received: %d jobs\n",amntRecv);
        hipHostFree(QinHost);
        hipHostFree(QoutHost);
        }
    else {
        assert(0);
    }
    double tf = get_time_msec();

    total_distance = distance_sqr_between_image_arrays(images_out, images_out_from_gpu);
    double avg_latency = 0;
    for (int i = 0; i < NREQUESTS; i++) {
        avg_latency += (req_t_end[i] - req_t_start[i]);
    }
    //printf("Total latency: %f\n",avg_latency); // REMOVE DEBUG
    avg_latency /= NREQUESTS;

    printf("mode = %s\n", mode == PROGRAM_MODE_STREAMS ? "streams" : "queue");
    printf("load = %lf (req/sec)\n", load);
    if (mode == PROGRAM_MODE_QUEUE) printf("threads = %d\n", threads_queue_mode);
    printf("distance from baseline %lf (should be zero)\n", total_distance);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (tf - ti) * 1e+3);
    printf("average latency = %lf (msec)\n", avg_latency);

    return 0;
}
